#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
static inline int roundup(int a, int b) {return ((a + b - 1) / b) * b;}
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x);
__global__ void preprocessor(float* raw, int ny, int nx, int nny) {
    int thread = threadIdx.x;
    int row = blockIdx.y;
    if (row >= ny) { return; }
    float* processed = raw + nx * ny;
    float mean = 0;
    float rootedSquaredSum = 0;

    for (int i = 0; i < nx; i += 64) {
        int col = i + thread;
        processed[nny * col + row] = (col < nx) ? raw[nx * row + col] : 0;
    }
}
__global__ void kernel(float* out, const float* in, int ny, int nx, int nny) {
    int ia = threadIdx.x;
    int ja = threadIdx.y;
    int ic = blockIdx.x;
    int jc = blockIdx.y;
    if (ic < jc || ic >= ny || jc >= ny){
        return;
    }
    const float* t = in + nx * ny;
    __shared__ float xx[4][64];
    __shared__ float yy[4][64];
    float v[8][8];
    for (int ib = 0; ib < 8; ++ib) {
        for (int jb = 0; jb < 8; ++jb) {
            v[ib][jb] = 0;
        }
    }
    for (int ks = 0; ks < nx; ks += 4) {
        int ija = ja * 8 + ia;
        int i = ic * 64 + ija;
        int j = jc * 64 + ija;
        for (int f = 0; f < 4; f++) {
            int k = ks + f;
            xx[f][ija] = t[nny * k + i];
            yy[f][ija] = t[nny * k + j];
        }
        __syncthreads();
        #pragma unroll
        for (int f = 0; f < 4; ++f) {
            float y[8];
            for (int jb = 0; jb < 8; ++jb) {
                y[jb] = yy[f][jb * 8 + ja];
            }
            for (int ib = 0; ib < 8; ++ib) {
                float x = xx[f][ib * 8 + ia];
                for (int jb = 0; jb < 8; ++jb) {
                    v[ib][jb] += x * y[jb];
                }
            }
        }
        __syncthreads();
    }
    for (int ib = 0; ib < 8; ++ib) {
        for (int jb = 0; jb < 8; ++jb) {
            int i = ic * 64 + ib * 8 + ia;
            int j = jc * 64 + jb * 8 + ja;
            if (i < ny && j < ny) {
                out[ny * j + i] = v[ib][jb];
            }
        }
    }
}
void correlate(int ny, int nx, const float *data, float *result) {

    int nnx = roundup(nx, 64);
    int nny = roundup(ny, 64);

    float* input = (float*) malloc(ny * nx * sizeof(float));
    for (int row = 0; row < ny; ++row) { // Getting the normalized input matrix
        float mean = 0; // Get the mean of the column
        float rootedSquaredSum = 0;
        for (int col = 0; col < nx; ++col) {
            mean += data[col + nx * row];
        }
        mean /= nx;
        for (int col = 0; col < nx; ++col) { // Get squared sum of the row
            float diff = data[col + nx * row] - mean;
            rootedSquaredSum += diff * diff;
            input[col + nx * row] = diff;
        }
        rootedSquaredSum = sqrt(rootedSquaredSum);
        for (int col = 0; col < nx; ++col) { // Normalize inputs
            input[col + nx * row] /= rootedSquaredSum;
        }
    }
    float* inGPU = NULL; // Initialize buffers
    float* outGPU = NULL;
    CHECK(hipMalloc((void**)&inGPU, (nny * nnx + ny * nx) * sizeof(float)));
    CHECK(hipMalloc((void**)&outGPU, nny * nny * sizeof(float)));
    CHECK(hipMemcpy(inGPU, input, ny * nx * sizeof(float), hipMemcpyHostToDevice)); // Move input to GPU
    free(input);
    { // Preprocessing
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, nny);
        preprocessor<<<dimGrid, dimBlock>>>(inGPU, ny, nx, nny);
        CHECK(hipGetLastError());
    }
    { // Compute
        dim3 dimBlock(8, 8);
        dim3 dimGrid(nny / 64, nny / 64);
        CHECK(hipMemset(outGPU, 0, nny * nny * sizeof(float)));
        kernel<<<dimGrid, dimBlock>>>(outGPU, inGPU, ny, nx, nny);
        CHECK(hipGetLastError());
    }
    CHECK(hipMemcpy(result, outGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost)); // Bring back output from GPU
    CHECK(hipFree(inGPU)); // Cleanup
    CHECK(hipFree(outGPU));
}