#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
static inline int divup(int a, int b) {return (a + b - 1)/b;}
static inline int roundup(int a, int b) {return divup(a, b) * b;}
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)

__global__ void preprocessor(const float* raw, float* processed, int ny, int nx, int nnx) {
    int thread = threadIdx.x;
    int block = blockIdx.y;

    for (int i = 0; i < nnx; i += 64) {
        int j = i + thread;
        processed[nnx * block + j] = (block < ny && j < nx) ? raw[nx * i + j] : 0;
    }
}

__global__ void kernel(float* out, const float* input, int ny, int nx) {
    int col = threadIdx.x + blockIdx.x * blockDim.x; // handling i, aka innerRow
    int row = threadIdx.y + blockIdx.y * blockDim.y; // handling j, aka outerRow
    if (row > col || row >= ny || col >= ny) {return;}
    else if (row == col) {
        out[col + row * ny] = 1;
        return;
    }
    float sum = 0;
    for (int n = 0; n < nx; ++n) {
        sum += input[n + col * nx] * input[n + row * nx];
    }
    out[col + row * ny] = sum;
}
void correlate(int ny, int nx, const float *data, float *result) {

    int nnx = roundup(nx, 64);
    int nny = roundup(ny, 64);

    float* input = (float*) malloc(nny * nnx * sizeof(float));
    for (int row = 0; row < ny; ++row) { // Getting the normalized input matrix
        float mean = 0; // Get the mean of the column
        float rootedSquaredSum = 0;
        for (int col = 0; col < nx; ++col) {
            mean += data[col + nx * row];
        }
        mean /= nx;
        for (int col = 0; col < nx; ++col) { // Get squared sum of the row
            float diff = data[col + nx * row] - mean;
            rootedSquaredSum += diff * diff;
            input[col + nx * row] = diff;
        }
        rootedSquaredSum = sqrt(rootedSquaredSum);
        for (int col = 0; col < nx; ++col) { // Normalize inputs
            input[col + nx * row] /= rootedSquaredSum;
        }
    }
    float* rawGPU = NULL;
    float* inGPU = NULL; // Initialize buffers
    float* outGPU = NULL;
    CHECK(hipMalloc((void**)&rawGPU, ny * nx * sizeof(float)));
    CHECK(hipMalloc((void**)&inGPU, nny * nnx * sizeof(float)));
    CHECK(hipMalloc((void**)&outGPU, nny * nny * sizeof(float)));
    CHECK(hipMemcpy(rawGPU, input, ny * nx * sizeof(float), hipMemcpyHostToDevice)); // Move input to GPU

    { // Preprocessing
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, nny);
        preprocessor<<<dimGrid, dimBlock>>>(rawGpu, inGpu, ny, nx, nnx);
    }

    {
        dim3 dimBlock(8, 8);
        dim3 dimGrid(nnx / 64, nny / 64);
        
    }
    
    dim3 dimBlock(16, 16); // Kernel preparation
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));

    CHECK(hipMemset(outGPU, 0, nny * nny * sizeof(float)));
    kernel<<<dimGrid, dimBlock>>>(outGPU, inGPU, ny, nx); // Run kernel
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(result, outGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost)); // Bring back output from GPU
    CHECK(hipFree(inGPU)); // Cleanup
    CHECK(hipFree(outGPU));
    free(input);
}