#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
static inline int divup(int a, int b) {return (a + b - 1)/b;}
static inline int roundup(int a, int b) {return divup(a, b) * b;}
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)
// __global__ void preprocessor(int ny, int nx, int nn, const float* originalData, const float* processedData) {
//     int ti = threadIdx.x;
//     int bi = blockIdx.y;
// }
__global__ void kernel(float* out, const float* input, int ny, int nx) {
    int col = threadIdx.x + blockIdx.x * blockDim.x; // handling i, aka innerRow
    int row = threadIdx.y + blockIdx.y * blockDim.y; // handling j, aka outerRow
    if (row > col || row >= ny || col >= ny) {return;}
    else if (row == col) {
        out[col + row * ny] = 1;
        return;
    }
    float sum = 0;
    for (int n = 0; n < nx; ++n) {
        sum += input[n + col * nx] * input[n + row * nx];
    }
    out[col + row * ny] = sum;
}
void correlate(int ny, int nx, const float *data, float *result) {
    float* input = (float*) malloc(ny * nx * sizeof(float));
    for (int row = 0; row < ny; ++row) { // Getting the normalized input matrix
        float mean = 0; // Get the mean of the column
        float rootedSquaredSum = 0;
        for (int col = 0; col < nx; ++col) {
            mean += data[col + nx * row];
        }
        mean /= nx;
        for (int col = 0; col < nx; ++col) { // Get squared sum of the row
            float diff = data[col + nx * row] - mean;
            rootedSquaredSum += diff * diff;
            input[col + nx * row] = diff;
        }
        rootedSquaredSum = sqrt(rootedSquaredSum);
        for (int col = 0; col < nx; ++col) { // Normalize inputs
            input[col + nx * row] /= rootedSquaredSum;
        }
    }
    float* inGPU = NULL; // Initialize buffers
    float* outGPU = NULL;
    CHECK(hipMalloc((void**)&inGPU, ny * nx * sizeof(float)));
    CHECK(hipMalloc((void**)&outGPU, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(inGPU, input, ny * nx * sizeof(float), hipMemcpyHostToDevice)); // Move input to GPU
    dim3 dimBlock(16, 16); // Kernel preparation
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
    CHECK(hipMemset(outGPU, 0, ny * ny * sizeof(float)));
    kernel<<<dimGrid, dimBlock>>>(outGPU, inGPU, ny, nx); // Run kernel
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(result, outGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost)); // Bring back output from GPU
    CHECK(hipFree(inGPU)); // Cleanup
    CHECK(hipFree(outGPU));
    free(input);
}