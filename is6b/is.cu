
#include <hip/hip_runtime.h>
#include <new>
#include <cstdlib>
#include <algorithm>
#include <limits>
#include <iostream>
#include <immintrin.h>
typedef float float8_t __attribute__ ((vector_size (8 * sizeof(float))));
struct Result {
    int y0;
    int x0;
    int y1;
    int x1;
    float outer[3];
    float inner[3];
};
Result segment(int ny, int nx, const float *data) {
    int* sums = (int*) calloc((ny + 1) * (nx + 1), sizeof(int));
    for (int x = 1; x <= nx; ++x) {
        int lastSum = 0;
        for (int y = 1; y <= ny; ++y) {
            int newSum = 0;
            for (int k = 0; k < x; ++k) {
                newSum += data[k * 3 + (y - 1) * 3 * nx];
            }
            lastSum += newSum;
            sums[x + y * (nx + 1)] = lastSum;
        }
    }
    int totalElems = ny * nx;
    int totalSum = sums[nx + ny * (nx + 1)];
    float bestError = std::numeric_limits<float>::max();
    Result best;
    for (int height = 1; height <= ny; ++height) {
        for (int width = 1; width <= nx; ++width) {
            for (int y0 = 0; y0 <= ny - height; ++y0) {
                for (int x0 = 0; x0 <= nx - width; ++x0) {
                    Result result{0, 0, 0, 0, {0, 0, 0}, {0, 0, 0}};
                    float localError = std::numeric_limits<float>::max();
                    int elemsInWindow = height * width;
                    float invElemsInWindow = 1.0 / elemsInWindow;
                    int elemsInBg = totalElems - elemsInWindow;
                    float invElemsInBg = 1.0 / elemsInBg;
                    int x1 = x0 + width;
                    int y1 = y0 + height;
                    int windowSum = sums[x1 + y1 * (nx + 1)] - sums[x1 + y0 * (nx + 1)] - sums[x0 + y1 * (nx + 1)] + sums[x0 + y0 * (nx + 1)];
                    int bgSum = totalSum - windowSum;
                    float newError = - (invElemsInBg * bgSum * bgSum + invElemsInWindow * windowSum * windowSum);
                    if (newError < localError) {
                        localError = newError;
                        result.y0 = y0;
                        result.x0 = x0;
                        result.y1 = y1;
                        result.x1 = x1;
                        float avgWindowColor = invElemsInWindow * windowSum;
                        float avgBgColor = invElemsInBg * bgSum;
                        for (int c = 0; c < 3; ++c) {
                            result.inner[c] = avgWindowColor;
                            result.outer[c] = avgBgColor;
                        }
                    }
                }
            }
        }
        // if (localError < bestError) {
        //     best = result;
        //     bestError = localError;
        // }
    }
    free(sums);
    return best;
}
