#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
static inline int divup(int a, int b) {return (a + b - 1)/b;}
static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
#define CHECK(x) check(x, #x)
__global__ void kernel(float* out, const float* in, int ny, int nx, const double* prep) {
    int col = threadIdx.x + blockIdx.x * blockDim.x; // handling i, aka innerRow
    int row = threadIdx.y + blockIdx.y * blockDim.y; // handling j, aka outerRow
    if (row > col || row >= ny || col >= ny) {
        return;
    }
    if (row == col) {
        out[col + row * ny] = 1;
        return;
    }
    double sumIJ = 0;
    for (int n = 0; n < nx; ++n) {
        sumIJ += double(in[n + col * nx]) * in[n + row * nx];
    }
    out[col + row * ny] = (sumIJ * nx - prep[col] * prep[row]) 
        / sqrt((prep[col + ny] * nx - prep[col] * prep[col]) * (prep[row + ny] * nx - prep[row] * prep[row]));
}
void correlate(int ny, int nx, const float *data, float *result) {
    double prep[2*ny]; // Precalculating the sums and squared sums per row
    for (int r = 0; r < ny; ++r) {
        prep[r] = 0.0;
        prep[r + ny] = 0.0;
        for (int c = 0; c < nx; ++c) {
            prep[r] += double(data[c + r * nx]);
            prep[r + ny] += double(data[c + r * nx]) * data[c + r * nx];
        }
    }
    double* prepGPU = NULL; // Intialize GPU buffers
    CHECK(hipMalloc((void**)&prepGPU, 2 * ny * sizeof(double)));
    float* inGPU = NULL; 
    CHECK(hipMalloc((void**)&inGPU, ny * nx * sizeof(float)));
    float* outGPU = NULL;
    CHECK(hipMalloc((void**)&outGPU, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(inGPU, data, ny * nx * sizeof(float), hipMemcpyHostToDevice)); // Move input to GPU
    CHECK(hipMemcpy(prepGPU, prep, 2 * ny * sizeof(double), hipMemcpyHostToDevice));
    dim3 dimBlock(16, 16); // Kernel preparation
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
    CHECK(hipMemset(outGPU, 0, ny * ny * sizeof(float)));
    kernel<<<dimGrid, dimBlock>>>(outGPU, inGPU, ny, nx, prepGPU); // Run kernel
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(result, outGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost)); // Bring back output from GPU
    CHECK(hipFree(inGPU)); // Cleanup
    CHECK(hipFree(outGPU));
    CHECK(hipFree(prepGPU));
}